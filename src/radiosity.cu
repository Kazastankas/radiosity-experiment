#include "hip/hip_runtime.h"

#include "radiosity.hpp"
#include "cutil_math.hpp"
#include <stdio.h>
#include <limits>

namespace radiosity {

#define PI 3.14159265358979f

float visible(Scene* scene, size_t src, size_t dst) 
{

  float3 pos = scene->patches[src].corner_pos;
  float3 dir = scene->patches[dst].corner_pos -
               scene->patches[src].corner_pos;
  float hit_time = length(dir);
  dir = normalize(dir);

  // Check versus everything else, see if something else gets hit first.
  for (size_t i = 0; i < scene->patches.size(); i++) {
    if (i == src || i == dst) continue;

    float3 p0 = scene->patches[i].corner_pos;
    float3 p1 = p0 + scene->patches[i].x_vec;
    float3 p2 = p0 + scene->patches[i].y_vec;
    
    // column vectors
    float3 c1 = -dir;
    float3 c2 = p1 - p0;
    float3 c3 = p2 - p0;
    float det = dot(c1, cross(c2, c3));
    
    // inverted row vectors
    float3 r1 = cross(c2, c3) / det;
    float3 r2 = cross(c3, c1) / det;
    float3 r3 = cross(c1, c2) / det;
    
    // coefficients
    float t_hit = dot(r1, pos - p0);
    float u_hit = dot(r2, pos - p0);
    float v_hit = dot(r3, pos - p0);
    
    // We've hit something else first, abort!
    if (t_hit > 0 && t_hit < hit_time &&
        u_hit >= scene->patches[i].x_min && u_hit <= scene->patches[i].x_max &&
        v_hit >= scene->patches[i].y_min && v_hit <= scene->patches[i].y_max)
    {
      return false;
    }
  }

  return true;
}


bool calc_radiosity(Scene* scene, float* matrix, size_t dim)
{
  //Populate energy-transfer matrix
  for (size_t x = 0; x < dim; x++) {
    if (x % 32 == 0) {
      printf("Starting row %d\n", x);
    }
    for (size_t y = 0; y < x; y++) {
      bool v = visible(scene, x, y);
      if (!v) {
        matrix[y * dim + x] = matrix[x * dim + y] = 0.0f;
        continue;
      }

      float ff = form_factor(&scene->patches[y], &scene->patches[x]);
      matrix[y * dim + x] = -ff * scene->patches[y].reflectance;
      matrix[x * dim + y] = -ff * scene->patches[x].reflectance;
    }
    matrix[x * dim + x] = 1.0f;
  }
  //Populate initial state
  float *energies = new float[dim];
  float *sol_0    = new float[dim];
  float *sol_1    = new float[dim];
  for(size_t ii = 0; ii < dim; ii++)
  {
    energies[ii] = scene->patches[ii].emission;
    sol_0[ii] = energies[ii];
    sol_1[ii] = energies[ii];
  }
  
  //Solve, then populate textures
  solve_radiosity(matrix, energies, sol_0, sol_1, dim);

  //Populate patches with solved colors
  for(size_t x = 0; x < dim; x++)
  {
    //printf("%f ", sol_1[x]);
    scene->patches[x].color = sol_1[x] * scene->patches[x].color;
  }
  printf("\n");
  return true;
}

//Calculate the form factor between two planes
float form_factor(Plane *p1, Plane *p2)
{
	float3 p1_norm = cross(p1->x_vec, p1->y_vec);
	float3 p2_norm = cross(p2->x_vec, p2->y_vec);
	float a1 = length(p1_norm);
	float a2 = length(p2_norm);

	float3 btwn = (p1->corner_pos + 0.5 * p1->x_vec + 0.5 * p1->y_vec) -
				  (p2->corner_pos + 0.5 * p2->x_vec + 0.5 * p2->y_vec);
	float  dist = length(btwn);

	btwn    = normalize(btwn);
	p1_norm = normalize(p1_norm);
	p2_norm = normalize(p2_norm);

	float dTheta = dot(btwn, p1_norm) * dot(btwn, p2_norm);
	// since we effectively divide by a1 at the end, only take on a2
  float dArea  = a2;
	float ff = dTheta * dArea / (dist * dist * PI);

  return ff;
}

__host__ //__device__
void jacobi(size_t ii, float *x_0, float *x_1, float *M, float* b, size_t dim)
{
	float acc = 0;

	  
	for(size_t jj = 0; jj < dim; jj++)
	{
    if (ii == jj) continue;
		acc += M[ii*dim + jj] * x_0[jj];
    //printf("(%d,%d)%f*%f ", jj, ii, M[ii*dim + jj], x_0[jj]);
  }

  //printf("\naccum for x = %d is %f\n", ii, acc);
	x_1[ii] = b[ii] - acc;  // (b[ii]- acc) / M[ii*dim + ii];
}

__global__
void jacobi_GPU(float *x_0, float *x_1, float *M, float *b, size_t dim)
{
	size_t ii = blockIdx.x * blockDim.x + threadIdx.x;

	//Check index
	if(ii >= dim)
		return;

//	jacobi(ii, x_0, x_1, M, b, dim);
}

void jacobi_CPU(float *x_0, float *x_1, float *M, float *b, size_t dim)
{
	for(size_t ii = 0; ii < dim; ii++)
	{
		jacobi(ii, x_0, x_1, M, b, dim);
	}
}

__host__
void solve_radiosity(float *M, float *b, float *sol_0, float *sol_1, size_t dim)
{
	size_t iters = 100;

	for(size_t ii = 0; ii < iters; ii++)
	{
		jacobi_CPU(sol_0, sol_1, M, b, dim);
		jacobi_CPU(sol_1, sol_0, M, b, dim);
		//jacobi_GPU<<<>>>(x_0, x_1, M, b, dim);
		//jacobi_GPU<<<>>>(x_1, x_0, M, b, dim);
	}
}

}
