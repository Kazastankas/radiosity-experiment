#include "hip/hip_runtime.h"

#include "radiosity.hpp"
#include "cutil_math.hpp"
#include <stdio.h>
#include <limits>

namespace radiosity {

#define PI 3.14159265358979f


bool initialize_radiosity(Scene* scene)
{
  return true;
  // TODO implement
};

void render_image(uint8_t* color, size_t width, size_t height,
                  const Scene* scene, const Camera* camera)
{
  float3 v, w, u;
  float l, r, b, t, n;
  float fovrad = camera->fov;
  
  v = camera->up;
  w = -camera->dir;
  u = cross(v, w);
  
  n = 1;
  t = tan(fovrad / 2.0f) * n;
  b = -t;
  l = b * camera->aspect_ratio;
  r = -l;
 
  for (size_t i = 0; i < width; i++) {
    for (size_t j = 0; j < height; j++) {
      float u_s = l + (r - l) * i / width;
      float v_s = b + (t - b) * j / height;
      float w_s = -n;
      
      float3 vray = normalize(u_s * u + v_s * v + w_s * w);
      trace_ray(color, i, j, camera->pos, vray, scene, height);
    }
  }
}

void trace_ray(uint8_t* color, size_t x, size_t y, float3 pos, float3 dir,
               const Scene* scene, size_t height)
{
  size_t idx = 4 * (y * height + x);
  float hit_time = std::numeric_limits<float>::infinity();
  
  // reset
  color[idx] = color[idx + 1] = color[idx + 2] = color[idx + 3] = 0;

  // Find the closest thing hit.
  for (size_t i = 0; i < scene->patches.size(); i++) {
    float3 p0 = scene->patches[i].corner_pos;
    float3 p1 = p0 + scene->patches[i].x_vec;
    float3 p2 = p0 + scene->patches[i].y_vec;
    
    // column vectors
    float3 c1 = -dir;
    float3 c2 = p1 - p0;
    float3 c3 = p2 - p0;
    float det = dot(c1, cross(c2, c3));
    
    // inverted row vectors
    float3 r1 = cross(c2, c3) / det;
    float3 r2 = cross(c3, c1) / det;
    float3 r3 = cross(c1, c2) / det;
    
    // coefficients
    float t_hit = dot(r1, pos - p0);
    float u_hit = dot(r2, pos - p0);
    float v_hit = dot(r3, pos - p0);
    
    if (t_hit > 0 && t_hit < hit_time &&
        u_hit >= scene->patches[i].x_min && u_hit <= scene->patches[i].x_max &&
        v_hit >= scene->patches[i].y_min && v_hit <= scene->patches[i].y_max)
    {
      hit_time = t_hit;
      color[idx] = scene->patches[i].color.x * 255;
      color[idx + 1] = scene->patches[i].color.y * 255;
      color[idx + 2] = scene->patches[i].color.z * 255;
      color[idx + 3] = 255;
    }
  }
}

//Calculate the form factor between two planes
float form_factor(Plane *p1, Plane *p2)
{
	float3 p1_norm = cross(p1->x_vec, p1->y_vec);
	float3 p2_norm = cross(p2->x_vec, p2->y_vec);
	float a1 = length(p1_norm);
	float a2 = length(p2_norm);

	float3 btwn = p1->corner_pos - p2->corner_pos;
	float  dist = length(btwn);

	btwn    = normalize(btwn);
	p1_norm = normalize(p1_norm);
	p2_norm = normalize(p2_norm);

	float dTheta = dot(btwn, p1_norm) * dot(btwn, p2_norm);
	float dArea  = a1*a2;
	float ff = dTheta * dArea / (dist * dist * PI);

	return ff;
}

__host__ __device__
void jacobi(size_t ii, float *x_0, float *x_1, float *M, float* b)
{
	float acc = 0;
	size_t dim = 10; //TODO: matrix dimensions

	for(size_t jj = 0; jj < dim; jj++)
	{
		acc += M[ii*dim + jj] * x_0[jj];
	}

	x_1[ii] = (b[ii] - acc) / M[ii*dim + ii];
}

__global__
void jacobi_GPU(float *x_0, float *x_1, float *M, float *b)
{
	size_t ii = blockIdx.x * blockDim.x + threadIdx.x;

	//Check index
	size_t dim = 10;
	if(ii >= dim)
		return;

	jacobi(ii, x_0, x_1, M, b);
}

void jacobi_CPU(float *x_0, float *x_1, float *M, float *b)
{
	size_t dim = 10; //TODO: matrix dimensions
	for(size_t ii = 0; ii < dim; ii++)
	{
		jacobi(ii, x_0, x_1, M, b);
	}
}

__host__
void solveRadiosity()
{
	size_t iters = 5;
	float *M;
	float *x_0;
	float *x_1;
	float *b;

	for(size_t ii = 0; ii < iters; ii++)
	{
		jacobi_CPU(x_0, x_1, M, b);
		jacobi_CPU(x_1, x_0, M, b);
		//jacobi_GPU<<<>>>(x_0, x_1, M, b);
		//jacobi_GPU<<<>>>(x_1, x_0, M, b);
	}
}

}
